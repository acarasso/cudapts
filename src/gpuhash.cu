#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2014 David G. Andersen
 * This code is licensed under the Apache 2.0 license and may be used or re-used
 * in accordance with its terms.
 */
/**
 * When building in SM35 Uncomment the following line.  NVCC is apparently retarded when it comes to
 * int64's and can't figure out how to use new instructions for rotates
 **/

#define __SM_35__
//#include <inttypes.h>
#include <stdio.h>
#include "gpuhash.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <sm_35_intrinsics.h>
//#include <thrust/sort.h>
__device__ void sha512_block(uint64_t H[8], const uint64_t data[5]);
__global__ void search_sha512_kernel(const  uint64_t *  __restrict dev_data,  uint64_t * __restrict dev_hashes,  uint32_t * __restrict dev_countbits);
__global__ void filter_sha512_kernel( uint64_t * __restrict dev_hashes, const  uint32_t * __restrict dev_countbits);
__global__ void filter_and_rewrite_sha512_kernel( uint64_t * __restrict dev_hashes, const  uint32_t * __restrict dev_countbits,  uint64_t *  __restrict dev_results);
__global__ void populate_filter_kernel( uint64_t * __restrict dev_hashes,  uint32_t *__restrict dev_countbits);



#define SWAP64(n) \
  (((n) << 56)                                        \
   | (((n) & 0xff00) << 40)                        \
   | (((n) & 0xff0000) << 24)                        \
   | (((n) & 0xff000000) << 8)                        \
   | (((n) >> 8) & 0xff000000)                        \
   | (((n) >> 24) & 0xff0000)                        \
   | (((n) >> 40) & 0xff00)                        \
   | ((n) >> 56))


/* Empty constructor, please call Initialize */
GPUHasher::GPUHasher(int gpu_device_id) {
  device_id = gpu_device_id;
}

int GPUHasher::Initialize() {
  hipError_t error;
  
  error = hipSetDevice(device_id);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not attach to CUDA device %d: %d\n", device_id, error);
    exit(-1);
  }

  hipStream_t *streamptr = (hipStream_t *)opaqueStream_t;
  error = hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

  size_t free, total;
  hipMemGetInfo(&free, &total);
  printf("Initializing.  Device has %ld free of %ld total bytes of memory\n", free, total);

  error = hipMalloc((void **)&dev_data, sizeof(uint64_t)*16);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  hipStreamCreate(streamptr);

#define MOMENTUM_N_HASHES (1<<26)
  /* Note:  This is the allocation size.  We can only use
   * one less than this because each countbit entry uses two bits. */
#define NUM_COUNTBITS_POWER 31
#define COUNTBITS_SLOTS_POWER (NUM_COUNTBITS_POWER-1)
#define NUM_COUNTBITS_WORDS (1<<(NUM_COUNTBITS_POWER-5))
  
  error = hipMalloc((void **)&dev_hashes, sizeof(uint64_t)*MOMENTUM_N_HASHES);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    return -1;
  }

  error = hipMalloc((void **)&dev_countbits, sizeof(uint32_t)*NUM_COUNTBITS_WORDS);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  /* Results holds any maybe-colliding keys */
  error = hipMalloc((void **)&dev_results, sizeof(uint64_t)*GPUHasher::N_RESULTS);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not malloc dev_data (%d)\n", error);
    exit(-1);
    return -1;
  }

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(search_sha512_kernel), hipFuncCachePreferL1);

  return 0;

}

GPUHasher::~GPUHasher() {
  if (dev_hashes != NULL) { hipFree(dev_hashes); }
  if (dev_data != NULL) { hipFree(dev_data); }
}

int GPUHasher::ComputeHashes(uint64_t data[16], uint64_t *hashes) {
  hipError_t error;
  hipStream_t *streamptr = (hipStream_t *)opaqueStream_t;
  error = hipMemcpy(dev_data, data, sizeof(uint64_t)*16, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    fprintf(stderr, "Could not memcpy dev_data (%d)\n", error);
    return -1;
  }

  // I want:  64 threads per block
  // 128 blocks per grid entry
  // 1024 grid slots

  dim3 gridsize(4096,32);
  hipMemsetAsync(dev_results, 0, sizeof(uint64_t)*N_RESULTS, *streamptr);
  hipMemsetAsync(dev_countbits, 0, sizeof(uint32_t)*NUM_COUNTBITS_WORDS, *streamptr);
  search_sha512_kernel<<<gridsize, 64, 0, *streamptr>>>(dev_data, dev_hashes, dev_countbits);
  filter_sha512_kernel<<<gridsize, 64, 0, *streamptr>>>(dev_hashes, dev_countbits);
  hipMemsetAsync(dev_countbits, 0, sizeof(uint32_t)*NUM_COUNTBITS_WORDS, *streamptr);
  populate_filter_kernel<<<gridsize, 64, 0, *streamptr>>>(dev_hashes, dev_countbits);
  filter_and_rewrite_sha512_kernel<<<gridsize, 64, 0, *streamptr>>>(dev_hashes, dev_countbits, dev_results);
  error = hipMemcpyAsync(hashes, dev_results, sizeof(uint64_t)*N_RESULTS, hipMemcpyDeviceToHost, *streamptr);

  error = hipDeviceSynchronize();
  if (error != hipSuccess) {
    fprintf(stderr, "Error in kernel exec (%d)\n", error);
    return -1;
  }

  if (error != hipSuccess) {
    fprintf(stderr, "Could not memcpy dev_hashes out (%d)\n", error);
    return -1;
  }
  return 0;
}

#define SHA512_HASH_WORDS 8 /* 64 bit words */

__constant__ const uint64_t iv512[SHA512_HASH_WORDS] = {
  0x6a09e667f3bcc908LL,
  0xbb67ae8584caa73bLL,
  0x3c6ef372fe94f82bLL,
  0xa54ff53a5f1d36f1LL,
  0x510e527fade682d1LL,
  0x9b05688c2b3e6c1fLL,
  0x1f83d9abfb41bd6bLL,
  0x5be0cd19137e2179LL
};

__device__
void set_or_double( uint32_t * __restrict__ countbits, uint32_t whichbit) {
  /* Kind of like a saturating add of two bit values.
   * First set is 00 -> 01.  Second set is 01 -> 11
   * Beyond that stays 11
   */
  uint32_t whichword = whichbit/16;
  uint32_t bitpat = 1UL << (2*(whichbit%16));
  uint32_t old = atomicOr(&countbits[whichword], bitpat);
  if (old & bitpat) {
    uint32_t secondbit = (1UL<<((2*(whichbit%16)) +1));
    if (!(old & secondbit)) {
      atomicOr(&countbits[whichword], secondbit);
    }
  }
}

__device__ inline
void add_to_filter( uint32_t * __restrict__ countbits, const uint64_t hash) {
  uint32_t whichbit = (uint32_t(hash>>14) & ((1UL<<COUNTBITS_SLOTS_POWER)-1));
  set_or_double(countbits, whichbit);
}

__device__ inline
bool is_in_filter_twice(const  uint32_t * __restrict__ countbits, const uint64_t hash) {
  uint32_t whichbit = (uint32_t(hash>>14) & ((1UL<<COUNTBITS_SLOTS_POWER)-1));
  uint32_t cbits = countbits[whichbit/16];
  
  return (cbits & (1UL<<((2*(whichbit%16))+1)));
}


__global__
void search_sha512_kernel(const  uint64_t * __restrict__ dev_data,  uint64_t * __restrict__ dev_hashes,  uint32_t * __restrict__ dev_countbits) {
  uint64_t H[8];
  uint64_t D[5];
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 5; i++) {
    D[i] = dev_data[i]; /* constant memory would be better */
  }

  D[0] = (D[0] & 0xffffffff00000000) | (spot*8);
  for (int i = 1; i < 5; i++) {
    D[i] = SWAP64(D[i]);
  }

  sha512_block(H, D);

  for (int i = 0; i < 8; i++) {
    add_to_filter(dev_countbits, H[i]);
#define POOLSIZE (1<<23)
    dev_hashes[i*POOLSIZE+spot] = H[i];
  }
}

__global__
void filter_sha512_kernel( uint64_t * __restrict__ dev_hashes, const  uint32_t * __restrict__ dev_countbits) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];
    bool c = is_in_filter_twice(dev_countbits, myword);
    if (!c) {
      dev_hashes[i*POOLSIZE+spot] = 0;
    }

  }
}


__global__
void populate_filter_kernel( uint64_t * __restrict__ dev_hashes,  uint32_t * __restrict__ dev_countbits) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];
    if (myword) {
      add_to_filter(dev_countbits, (myword>>18));
    }
  }
}

__global__
void filter_and_rewrite_sha512_kernel( uint64_t * __restrict__ dev_hashes, const  uint32_t * __restrict__ dev_countbits, uint64_t *  __restrict__ dev_results) {
  uint32_t spot = (((gridDim.x * blockIdx.y) + blockIdx.x)* blockDim.x) + threadIdx.x;
  for (int i = 0; i < 8; i++) {
    uint64_t myword = dev_hashes[i*POOLSIZE+spot];

    if (myword && is_in_filter_twice(dev_countbits, (myword>>18))) {
      uint32_t result_slot = atomicInc((uint32_t *)dev_results, GPUHasher::N_RESULTS);
      dev_results[result_slot*2+1] = (myword >> 14); /* the actual momentum val */
      dev_results[result_slot*2+2] = (spot*8+i);
    }
  }
}



/***** SHA 512 code is derived from Lukas Odzioba's sha512 crypt implementation within JohnTheRipper.  It has its own copyright */
/*
* This software is Copyright (c) 2011 Lukas Odzioba <lukas dot odzioba at gmail dot com>
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/

#define rol(x,n) ((x << n) | (x >> (64-n)))
#define ror(x,n) ((x >> n) | (x << (64-n)))

#ifdef __SM_35__


#define fror(x, n) (fastrotateright(x,n))
#define frol(x, n) (fastrotateleft(x,n))
#define tsplit(x) ((uint32_t *)&x)[0]
#define lsplit(x) ((uint32_t *)&x)[1]
#define bfror(x,n) (bigfror(x, 32, n-32)) 
 inline __device__ uint64_t fastrotateright(const uint64_t x, const uint32_t n)
{
	uint64_t out =0;
	tsplit(out) = __funnelshift_rc(tsplit(x),lsplit(x), n);
	lsplit(out) = __funnelshift_rc(lsplit(x), tsplit(x), n);
	return out;
}

inline __device__ uint64_t fastrotateleft(const uint64_t x, const uint32_t n)
{
	uint64_t out = 0;
	lsplit(out) = __funnelshift_lc(tsplit(x), lsplit(x), n);
	tsplit(out) = __funnelshift_lc(lsplit(x), tsplit(x), n);
	return out;
}


//inline __device__ uint64_t bigfror(const uint64_t x, const uint32_t n, const uint32_t m)
//{
//	uint64_t out =0;
//	uint32_t t =0;
//	tsplit(out) = __funnelshift_rc(tsplit(x),lsplit(x), 32);
//	lsplit(out) = __funnelshift_rc(lsplit(x), tsplit(x), 32);
//	t = __funnelshift_rc(tsplit(out),lsplit(out), m);
//	lsplit(out) = __funnelshift_rc(lsplit(out), tsplit(out), m);
//	tsplit(out) = t;
//	return out;
//}
#define Ch(x,y,z) ((x & y) ^ ( (~x) & z))
#define Maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))
#define Sigma0(x) ((fror(x,28))  ^ (frol(x,30)) ^ (frol(x,25)))
#define Sigma1(x) ((fror(x,14))  ^ (fror(x,18)) ^ (frol(x,23)))
#define sigma0(x) ((fror(x,1))  ^ (fror(x,8)) ^(x>>7))
#define sigma1(x) ((fror(x,19)) ^ (frol(x,3)) ^(x>>6))

#else
#define rol(x,n) ((x << n) | (x >> (64-n)))
#define ror(x,n) ((x >> n) | (x << (64-n)))

#define Ch(x,y,z) ((x & y) ^ ( (~x) & z))
#define Maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))
#define Sigma0(x) ((ror(x,28))  ^ (ror(x,34)) ^ (ror(x,39)))
#define Sigma1(x) ((ror(x,14))  ^ (ror(x,18)) ^ (ror(x,41)))
#define sigma0(x) ((ror(x,1))  ^ (ror(x,8)) ^(x>>7))
#define sigma1(x) ((ror(x,19)) ^ (ror(x,61)) ^(x>>6))
#endif
#define SWAP32(n) \
    (((n) << 24) | (((n) & 0xff00) << 8) | (((n) >> 8) & 0xff00) | ((n) >> 24))



__constant__ uint64_t k[] = {
	0x428a2f98d728ae22LL, 0x7137449123ef65cdLL, 0xb5c0fbcfec4d3b2fLL,
	    0xe9b5dba58189dbbcLL,
	0x3956c25bf348b538LL, 0x59f111f1b605d019LL, 0x923f82a4af194f9bLL,
	    0xab1c5ed5da6d8118LL,
	0xd807aa98a3030242LL, 0x12835b0145706fbeLL, 0x243185be4ee4b28cLL,
	    0x550c7dc3d5ffb4e2LL,
	0x72be5d74f27b896fLL, 0x80deb1fe3b1696b1LL, 0x9bdc06a725c71235LL,
	    0xc19bf174cf692694LL,
	0xe49b69c19ef14ad2LL, 0xefbe4786384f25e3LL, 0x0fc19dc68b8cd5b5LL,
	    0x240ca1cc77ac9c65LL,
	0x2de92c6f592b0275LL, 0x4a7484aa6ea6e483LL, 0x5cb0a9dcbd41fbd4LL,
	    0x76f988da831153b5LL,
	0x983e5152ee66dfabLL, 0xa831c66d2db43210LL, 0xb00327c898fb213fLL,
	    0xbf597fc7beef0ee4LL,
	0xc6e00bf33da88fc2LL, 0xd5a79147930aa725LL, 0x06ca6351e003826fLL,
	    0x142929670a0e6e70LL,
	0x27b70a8546d22ffcLL, 0x2e1b21385c26c926LL, 0x4d2c6dfc5ac42aedLL,
	    0x53380d139d95b3dfLL,
	0x650a73548baf63deLL, 0x766a0abb3c77b2a8LL, 0x81c2c92e47edaee6LL,
	    0x92722c851482353bLL,
	0xa2bfe8a14cf10364LL, 0xa81a664bbc423001LL, 0xc24b8b70d0f89791LL,
	    0xc76c51a30654be30LL,
	0xd192e819d6ef5218LL, 0xd69906245565a910LL, 0xf40e35855771202aLL,
	    0x106aa07032bbd1b8LL,
	0x19a4c116b8d2d0c8LL, 0x1e376c085141ab53LL, 0x2748774cdf8eeb99LL,
	    0x34b0bcb5e19b48a8LL,
	0x391c0cb3c5c95a63LL, 0x4ed8aa4ae3418acbLL, 0x5b9cca4f7763e373LL,
	    0x682e6ff3d6b2b8a3LL,
	0x748f82ee5defb2fcLL, 0x78a5636f43172f60LL, 0x84c87814a1f0ab72LL,
	    0x8cc702081a6439ecLL,
	0x90befffa23631e28LL, 0xa4506cebde82bde9LL, 0xbef9a3f7b2c67915LL,
	    0xc67178f2e372532bLL,
	0xca273eceea26619cLL, 0xd186b8c721c0c207LL, 0xeada7dd6cde0eb1eLL,
	    0xf57d4f7fee6ed178LL,
	0x06f067aa72176fbaLL, 0x0a637dc5a2c898a6LL, 0x113f9804bef90daeLL,
	    0x1b710b35131c471bLL,
	0x28db77f523047d84LL, 0x32caab7b40c72493LL, 0x3c9ebe0a15c9bebcLL,
	    0x431d67c49c100d4cLL,
	0x4cc5d4becb3e42b6LL, 0x597f299cfc657e2aLL, 0x5fcb6fab3ad6faecLL,
	    0x6c44198c4a475817LL,
};

__device__ void sha512_block(uint64_t H[8], const uint64_t data[5])
{
  uint64_t a = iv512[0];
  uint64_t b = iv512[1];
  uint64_t c = iv512[2];
  uint64_t d = iv512[3];
  uint64_t e = iv512[4];
  uint64_t f = iv512[5];
  uint64_t g = iv512[6];
  uint64_t h = iv512[7];

  uint64_t w[16];


  

	/* This can all be factored out onto the CPU setup, but let's
	 * get it working properly first. */
	/* n.b. - that optimizatoin of removing the swaps into setup
	 * will also work for our CPU version.  Just sayin' */
//#pragma unroll 16
	/* Lots of these middle entries are zero because of the pad */
        w[0] = SWAP64(data[0]);
#pragma unroll
	for (int i = 1; i < 5; i++)
		w[i] = data[i];
#pragma unroll
	for (int i = 5; i < 15; i++) {
	  w[i] = 0;
	}
	w[15] = 0x120; /* SWAP64(0x2001000000000000ULL); */

	uint64_t t1, t2;

	/* dga: Parts of this can be optimized for the first iteration
	 * to account for all of the fixed input values */

#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		t1 = k[i] + w[i] + h + Sigma1(e) + Ch(e, f, g);
		t2 = Maj(a, b, c) + Sigma0(a);

		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}

#pragma unroll
	for (int i = 16; i < 80; i++) {


		w[i & 15] =sigma1(w[(i - 2) & 15]) + sigma0(w[(i - 15) & 15]) + w[(i -16) & 15] + w[(i - 7) & 15];
		t1 = k[i] + w[i & 15] + h + Sigma1(e) + Ch(e, f, g);
		t2 = Maj(a, b, c) + Sigma0(a);

		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;

	}

        H[0] = iv512[0] + a;
	H[1] = iv512[1] + b;
	H[2] = iv512[2] + c;
	H[3] = iv512[3] + d;
	H[4] = iv512[4] + e;
	H[5] = iv512[5] + f;
	H[6] = iv512[6] + g;
	H[7] = iv512[7] + h;

#pragma unroll
	for (int i = 0; i < 8; i++) {
	  H[i] = (SWAP64(H[i]));
	}
}
